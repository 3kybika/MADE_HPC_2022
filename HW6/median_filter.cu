#include "hip/hip_runtime.h"
/*
Compiling:
    nvcc -arch sm_86 ./median_filter.cu -o median_filter
Using:
    ./median_filter <source_file_name> <target_file_name>
    source_file_name - path to source file, default value: default.jpg
    target_file_name - path to target file, default value: default_median_filter.jpg
*/ 

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define GRID_CNT 4
#define FILTER_SIZE 5 // WARNING! Must be odd!

__global__ void apply_median_filter(
    uint8_t* source_image,
    uint8_t* target_image,
    int width,
    int height,
    int size,
    int block_width,
    int block_height
) {

    int half_filter_size = FILTER_SIZE / 2;

    int block_start = (
        threadIdx.z * blockDim.x * blockDim.y + 
        threadIdx.y * blockDim.x + 
        threadIdx.x
    ) + (
        blockIdx.z * gridDim.x * gridDim.y +
        blockIdx.y * gridDim.x +
        blockIdx.x
    ) * blockDim.x * blockDim.y * blockDim.z;

    int height_block_idx =  block_start / (width / block_width);
    int width_block_idx =  block_start % (height / block_width);
    uint8_t filter_buffer[FILTER_SIZE * FILTER_SIZE];

    for (int i = 0; i < block_width; i++) {
        int x = block_width * width_block_idx + i;
        if (x > width - half_filter_size || x < half_filter_size)
            continue;

        for (int j = 0; j < block_height; j++) {
            int y = block_height* height_block_idx + j;
            if (y > height - half_filter_size || y < half_filter_size)
                continue;
 
            // fill filter
            for(int k = -half_filter_size; k <= half_filter_size; ++k) {
                for(int l = -half_filter_size; l <= half_filter_size; l++) {
                    filter_buffer[
                        half_filter_size + k + (half_filter_size + l) * FILTER_SIZE
                    ] = source_image[x + k + (y + l) * width];
                }
            }

            // apply filter
            for (int k = 0; k < FILTER_SIZE * FILTER_SIZE; ++k) {
                for (int l = 0; l < FILTER_SIZE * FILTER_SIZE - k; ++l) {
                    if (filter_buffer[l] < filter_buffer[l + 1]) {
                        uint8_t tmp = filter_buffer[l];
                        filter_buffer[l] =  filter_buffer[l + 1];
                        filter_buffer[l + 1] = tmp;
                    }
                }
            }
            target_image[x + y * width] = filter_buffer[(half_filter_size + 1) * (half_filter_size + 1)];
        }
    }
}


int main(int argc, char **argv) {

    // args
    char* source_path;
    char* target_path;

    if (argc == 2 && (
            strcmp(argv[1], "-h") == 0 ||
            strcmp(argv[1], "--help") == 0
        )
    ) {
        printf("App for blurring the image by median filter\n\n");
        printf("Compiling:\n");
        printf("    nvcc -arch sm_86 ./median_filter.cu -o median_filter\n");
        printf("Using:\n");
        printf("    ./median_filter <source_file_name> <target_file_name>\n");
        printf("    source_file_name - path to source file, default value: default.jpg\n");
        printf("    target_file_name - path to target file, default value: default_median_filter.jpg\n");

        return 0;
    }

    if (argc >= 2)
        source_path = argv[1];
    else
        source_path = (char*) "default.jpg";

    if (argc == 3)
        target_path = argv[2];
    else
        target_path = (char*) "default_median_filter.jpg";

    // source image
    int width;
    int height;
    int comp;
    uint8_t* source_image = stbi_load(source_path, &width, &height, &comp, 1);

    int image_size = height * width * sizeof(uint8_t);
    int block_width = width / GRID_CNT;
    int block_height = height / GRID_CNT;

    // cuda source image
    uint8_t* cuda_source_image;
    hipMalloc(&cuda_source_image, image_size);
    hipMemcpy(cuda_source_image, source_image, image_size, hipMemcpyHostToDevice);

    // target image
    uint8_t* target_image = (uint8_t*) malloc(image_size);

    // cuda target image
    uint8_t* cuda_target_image;
    hipMalloc(&cuda_target_image, image_size);

    // apply filter
    apply_median_filter<<<1, GRID_CNT * GRID_CNT>>>(
        cuda_source_image, cuda_target_image, width, height, image_size, block_width, block_height
    );
    hipDeviceSynchronize();

    // output image
    hipMemcpy(target_image, cuda_target_image, image_size, hipMemcpyDeviceToHost);
    stbi_write_png(target_path, width, height, 1, target_image, width);

    // free
    free(target_image);
    stbi_image_free(source_image);

    hipFree(cuda_source_image);
    hipFree(cuda_target_image);

    return 0;
}
